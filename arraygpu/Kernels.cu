#include "hip/hip_runtime.h"
//
// Created by chnlkw on 1/23/18.
//

#include "Kernels.h"

template<class T, class TOff>
__global__ void shuffle_by_idx_kernel(T *dst, const T *src, const TOff *idx, size_t size) {
    size_t i = blockDim.x * blockIdx.x + threadIdx.x;
    if (i < size)
        dst[i] = src[idx[i]];
};

template<class T, class TOff>
void shuffle_by_idx_gpu(T *dst, const T *src, const TOff *idx, size_t size, hipStream_t stream) {

    shuffle_by_idx_kernel << < (size + 31) / 32, 32, 0, stream >> > (dst, src, idx, size);
}

template void
shuffle_by_idx_gpu<float, size_t>(float *dst, const float *src, const size_t *idx, size_t size, hipStream_t stream);

template void
shuffle_by_idx_gpu<unsigned int, size_t>(unsigned int *dst, const unsigned int *src, const size_t *idx, size_t size,
                                         hipStream_t stream);

template void shuffle_by_idx_gpu<unsigned int, unsigned int>(unsigned int *, unsigned int const *, unsigned int const *,
                                                             unsigned long, hipStream_t);

template<class T>
__global__ void gpu_add_kernel(T *c, const T *a, const T *b, size_t size) { // c[i] = a[i] + b[i]
    size_t i = blockDim.x * blockIdx.x + threadIdx.x;
    if (i < size)
        c[i] = a[i] + b[i];
}

template<class T>
void gpu_add<>(T *c, const T *a, const T *b, size_t size, hipStream_t stream) { // c[i] = a[i] + b[i]
    gpu_add_kernel << < (size + 31) / 32, 32, 0, stream >> > (c, a, b, size);
}

template void gpu_add<int>(int *, const int *, const int *, size_t, hipStream_t stream);
